#include "hip/hip_runtime.h"
#include "Simulator.h"
#include "utils.h"

#include <iostream>
#include <hip/hip_runtime.h>

#include <thrust/device_new.h>

#define KERNEL_SCALE 4.0f
#define EPSILON 1e-30f

const bool DEBUG = false;

Simulator::Simulator() :particleSet(), scene(nullptr), kernel()
{
}


Simulator::~Simulator()
{
}

__global__ void initParticleSetPointer(
	DeviceParticleSetPointer* p,
	PCISPH::Vec3* position,
	PCISPH::Vec3* predictPosition,
	PCISPH::Vec3* velocity,
	PCISPH::Vec3* predictVelocity,
	PCISPH::Vec3* normal,
	PCISPH::Vec3* forces,
	PCISPH::Vec3* pressureForce,
	float* density,
	float* pressure,
	float particleMass,
	float maxDensityErr,
	size_t count
)
{
	p->position = position;
	p->predictPosition = predictPosition;
	p->velocity = velocity;
	p->predictVelocity = predictVelocity;
	p->normal = normal;
	p->forces = forces;
	p->pressureForce = pressureForce;
	p->density = density;
	p->pressure = pressure;
	p->particleMass = particleMass;
	p->maxDensityErr = maxDensityErr;
	p->count = count;
}

__global__ void initDeviceGrid(
	Grid* grid,
	float cellSize,
	PCISPH::Vec3 boxSize,
	PCISPH::uVec3 gridSize,
	size_t cellNumber
)
{
	grid->cellSize = cellSize;
	grid->boxSize = boxSize;
	grid->gridSize = gridSize;
	grid->cellNumber = cellNumber;
}

__global__
void initDeviceKernel(Kernel* k, float h)
{
	k->init(h);
}

void Simulator::init(const Scene *scene) {
	// Initialize scene
	this->scene = scene;

	// Initialize particleSet
	float particleMass = scene->referenceDensity * pow(2 * scene->particleRadius, 3);
	this->particleSet.init(particleMass);
	std::cout << "paricleMass = " << particleMass << std::endl;

	// Initialize fluid particles' and boundary particles' positions.
	this->initParticlesPositions();

	particleSetPointer = thrust::device_new<DeviceParticleSetPointer>();

	initParticleSetPointer << <1, 1 >> >(
		thrust::raw_pointer_cast(particleSetPointer),
		thrust::raw_pointer_cast(particleSet.position.data()),
		thrust::raw_pointer_cast(particleSet.predictPosition.data()),
		thrust::raw_pointer_cast(particleSet.velocity.data()),
		thrust::raw_pointer_cast(particleSet.predictVelocity.data()),
		thrust::raw_pointer_cast(particleSet.normal.data()),
		thrust::raw_pointer_cast(particleSet.forces.data()),
		thrust::raw_pointer_cast(particleSet.pressureForce.data()),
		thrust::raw_pointer_cast(particleSet.density.data()),
		thrust::raw_pointer_cast(particleSet.pressure.data()),
		particleSet.particleMass,
		particleSet.maxDensityErr,
		particleSet.count
		);

	// Initialize kernel
	float kernelRadius = KERNEL_SCALE * scene->particleRadius;
	kernel.init(kernelRadius);
	std::cout << "Kernel.H = " << kernel.H << std::endl;

	// Initialize grid
	this->fluidGrid.init(scene->boxSize, kernel.H);
	this->boundaryGrid.init(scene->boxSize, kernel.H);
	updateFluidGrid();


	// Initialize density variance scale
	this->initDensityVarianceScale();
	std::cout << "densityVarianceScale = " << this->densityVarianceScale << std::endl;

	offset.resize(fluidGrid.cellNumber + 1);


	d_fluidGrid = thrust::device_new<Grid>();
	initDeviceGrid << <1, 1 >> > (thrust::raw_pointer_cast(d_fluidGrid), fluidGrid.cellSize, fluidGrid.boxSize, fluidGrid.gridSize, fluidGrid.cellNumber);
	d_kernel = thrust::device_new<Kernel>();
	initDeviceKernel << <1, 1 >> > (thrust::raw_pointer_cast(d_kernel), kernelRadius);

	this->relax();

	std::cout << std::endl;
}
void Simulator::updateFluidGrid() {

	particleSet.h_velocity = particleSet.velocity;
	particleSet.h_position = particleSet.position;

	h_offset = offset;

	this->fluidGrid.update(particleSet.h_position, h_offset, [this](size_t i, size_t j) {
		std::swap(particleSet.h_position[i], particleSet.h_position[j]);
		std::swap(particleSet.h_velocity[i], particleSet.h_velocity[j]);
	});

	offset = h_offset;

	particleSet.position = particleSet.h_position;
	particleSet.velocity = particleSet.h_velocity;
}

__global__
void computeDensity(Grid* grid, DeviceParticleSetPointer* particles, size_t* offset, Kernel* kernel) {

	// compute fluid particles' densities
	for (size_t i = 0; i < particles->count; i++) {
		float fluidTerm = 0.0f;
		PCISPH::Vec3 pos = particles->position[i];

		glm::u64vec3 boundBoxMin = grid->getGridPos(pos - PCISPH::Vec3(grid->cellSize));
		glm::u64vec3 boundBoxMax = grid->getGridPos(pos + PCISPH::Vec3(grid->cellSize));

		for (auto z = boundBoxMin.z; z <= boundBoxMax.z; z++) {
			for (auto y = boundBoxMin.y; y <= boundBoxMax.y; y++) {
				for (auto x = boundBoxMin.x; x <= boundBoxMax.x; x++) {
					size_t cellIndex = grid->linearIndex(PCISPH::uVec3(x, y, z));
					for (size_t neighborIndex = offset[cellIndex]; neighborIndex < offset[cellIndex + 1]; neighborIndex++) {
						PCISPH::Vec3 r = pos - particles->position[neighborIndex];
						float rLen = PCISPH::length(r);
						if (rLen > kernel->H || rLen < EPSILON) return;
						fluidTerm += kernel->poly6Kernel(r);
					}
				}
			}
		}

		particles->density[i] = particles->particleMass * fluidTerm;
	}
}

__global__
void computeNormal(Grid* grid, DeviceParticleSetPointer* particles, size_t* offset, Kernel* kernel) {
	for (auto i = 0; i < particles->count; i++) {
		PCISPH::Vec3 n(0.0f);

		glm::u64vec3 boundBoxMin = grid->getGridPos(particles->position[i] - PCISPH::Vec3(grid->cellSize));
		glm::u64vec3 boundBoxMax = grid->getGridPos(particles->position[i] + PCISPH::Vec3(grid->cellSize));

		for (auto z = boundBoxMin.z; z <= boundBoxMax.z; z++) {
			for (auto y = boundBoxMin.y; y <= boundBoxMax.y; y++) {
				for (auto x = boundBoxMin.x; x <= boundBoxMax.x; x++) {
					size_t cellIndex = grid->linearIndex(PCISPH::uVec3(x, y, z));
					for (size_t neighborIndex = offset[cellIndex]; neighborIndex < offset[cellIndex + 1]; neighborIndex++) {
						PCISPH::Vec3 r = particles->position[i] - particles->position[neighborIndex];
						float rLen = PCISPH::length(r);
						if (rLen > kernel->H || rLen < EPSILON) return;
						n += kernel->poly6KernelGradient(r) / particles->density[neighborIndex];
					}
				}
			}
		}

		n *= kernel->H * particles->particleMass;
		particles->normal[i] = n;
	}
}

__global__
void computeForces(Grid* grid, DeviceParticleSetPointer* particleSet, size_t* offset, Kernel* kernel,
	float referenceDensity, float viscosityCoefficient, float surfaceTensionCoefficient, PCISPH::Vec3 gravity) {
	float squaredMass = particleSet->particleMass * particleSet->particleMass;

	for (auto i = 0; i < particleSet->count; i++) {
		PCISPH::Vec3 viscosity(0.f);
		PCISPH::Vec3 cohesion(0.f);
		PCISPH::Vec3 curvature(0.f);

		glm::u64vec3 boundBoxMin = grid->getGridPos(particleSet->position[i] - PCISPH::Vec3(grid->cellSize));
		glm::u64vec3 boundBoxMax = grid->getGridPos(particleSet->position[i] + PCISPH::Vec3(grid->cellSize));

		for (auto z = boundBoxMin.z; z <= boundBoxMax.z; z++) {
			for (auto y = boundBoxMin.y; y <= boundBoxMax.y; y++) {
				for (auto x = boundBoxMin.x; x <= boundBoxMax.x; x++) {
					size_t cellIndex = grid->linearIndex(PCISPH::uVec3(x, y, z));
					for (size_t j = offset[cellIndex]; j < offset[cellIndex + 1]; j++) {
						PCISPH::Vec3 r = particleSet->position[i] - particleSet->position[j];
						float rLen = PCISPH::length(r);
						if (rLen > kernel->H || rLen < EPSILON) return;

						PCISPH::Vec3 vDiff = particleSet->velocity[i] - particleSet->velocity[j];
						viscosity -= vDiff * kernel->viscosityKernelLaplacian(r) / particleSet->density[j];

						float Kij = 2.0f * referenceDensity / (particleSet->density[i] + particleSet->density[j]);
						cohesion += Kij * (r / rLen) * kernel->cohesionKernel(rLen);
						curvature += Kij * (particleSet->normal[i] - particleSet->normal[j]);
					}
				}
			}
		}



		viscosity *= viscosityCoefficient * squaredMass / particleSet->density[i];
		cohesion *= -surfaceTensionCoefficient * squaredMass;
		curvature *= -surfaceTensionCoefficient * particleSet->particleMass;

		particleSet->forces[i] = viscosity + cohesion + curvature + particleSet->particleMass * gravity;
	}
}

void Simulator::clearPressureAndPressureForce() {
	thrust::fill(particleSet.pressure.begin(), particleSet.pressure.end(), 0);
	thrust::fill(particleSet.pressureForce.begin(), particleSet.pressureForce.end(), PCISPH::Vec3(0, 0, 0));
}

__global__
void predictVelocityAndPosition(DeviceParticleSetPointer* particleSet, float timeStep) {
	for (int i = 0; i < particleSet->count; i++) {
		PCISPH::Vec3 acceleration = (particleSet->forces[i] + particleSet->pressureForce[i]) / particleSet->particleMass;
		particleSet->predictVelocity[i] = particleSet->velocity[i] + timeStep * acceleration;
		particleSet->predictPosition[i] = particleSet->position[i] + particleSet->predictVelocity[i] * timeStep;
	}
}

__global__
void updatePressure(Grid* grid, DeviceParticleSetPointer* particleSet, size_t* offset, Kernel* kernel, float referenceDensity, float densityVarianceScale) {

	for (int i = 0; i < particleSet->count; i++) {
		float fDensity = 0.f;

		glm::u64vec3 boundBoxMin = grid->getGridPos(particleSet->position[i] - PCISPH::Vec3(grid->cellSize));
		glm::u64vec3 boundBoxMax = grid->getGridPos(particleSet->position[i] + PCISPH::Vec3(grid->cellSize));

		for (auto z = boundBoxMin.z; z <= boundBoxMax.z; z++) {
			for (auto y = boundBoxMin.y; y <= boundBoxMax.y; y++) {
				for (auto x = boundBoxMin.x; x <= boundBoxMax.x; x++) {
					size_t cellIndex = grid->linearIndex(PCISPH::uVec3(x, y, z));
					for (size_t j = offset[cellIndex]; j < offset[cellIndex + 1]; j++) {
						PCISPH::Vec3 r = particleSet->predictPosition[i] - particleSet->predictPosition[j];
						float rLen = PCISPH::length(r);
						if (rLen > kernel->H || rLen < EPSILON) return;
						fDensity += kernel->poly6Kernel(r);
					}
				}
			}
		}

		float bDensity = 0.f;

		float density = fDensity * particleSet->particleMass + bDensity;
		float densityVariation = max(0.f, density - referenceDensity);
		particleSet->maxDensityErr = max(particleSet->maxDensityErr, densityVariation);

		particleSet->pressure[i] += densityVarianceScale * densityVariation;
	}
}

__global__
void updatePressureForce(Grid* grid, DeviceParticleSetPointer* particleSet, size_t* offset, Kernel* kernel) {
	for (int i = 0; i < particleSet->count; i++) {
		PCISPH::Vec3 pressureForce(0.f);

		glm::u64vec3 boundBoxMin = grid->getGridPos(particleSet->position[i] - PCISPH::Vec3(grid->cellSize));
		glm::u64vec3 boundBoxMax = grid->getGridPos(particleSet->position[i] + PCISPH::Vec3(grid->cellSize));

		for (auto z = boundBoxMin.z; z <= boundBoxMax.z; z++) {
			for (auto y = boundBoxMin.y; y <= boundBoxMax.y; y++) {
				for (auto x = boundBoxMin.x; x <= boundBoxMax.x; x++) {
					size_t cellIndex = grid->linearIndex(PCISPH::uVec3(x, y, z));
					for (size_t j = offset[cellIndex]; j < offset[cellIndex + 1]; j++) {
						PCISPH::Vec3 r = particleSet->predictPosition[i] - particleSet->predictPosition[j];
						float rLen = PCISPH::length(r);
						if (rLen > kernel->H || rLen < 1e-5) return;
						float term1 = particleSet->pressure[i] / particleSet->density[i] / particleSet->density[i];
						float term2 = particleSet->pressure[j] / particleSet->density[j] / particleSet->density[j];
						pressureForce -= particleSet->particleMass * (term1 + term2) * kernel->spikyKernelGradient(r);
					}
				}
			}
		}

		pressureForce *= particleSet->particleMass;
		particleSet->pressureForce[i] = pressureForce;
	}
}

__global__
void updateVelocityAndPosition(DeviceParticleSetPointer* particleSet, float timeStep) {
	for (int i = 0; i < particleSet->count; i++) {
		PCISPH::Vec3 acceleration = (particleSet->forces[i] + particleSet->pressureForce[i]) / particleSet->particleMass;
		particleSet->velocity[i] += acceleration * timeStep;
		particleSet->position[i] += particleSet->velocity[i] * timeStep;
	}
}

void Simulator::updateScene() {
	// TODO
}

void Simulator::relax() {
	update(10000);

	thrust::fill(particleSet.velocity.begin(), particleSet.velocity.end(), scene->initVelocity);
}

__device__
void collisionFunc(DeviceParticleSetPointer* particleSet, float restitution, size_t i, const PCISPH::Vec3 &n, const float d)
{
	particleSet->position[i] += d * n;
	particleSet->velocity[i] -= (1 + restitution) * PCISPH::dot(particleSet->velocity[i], n) * n;
}


__global__
void handleCollision(DeviceParticleSetPointer* particleSet, float restitution, PCISPH::Vec3 box) {

	for (int i = 0; i < particleSet->count; i++) {
		const auto &p = particleSet->position[i];
		PCISPH::Vec3 n;
		if (p.x < 0) {
			collisionFunc(particleSet, restitution, i, PCISPH::Vec3(1.f, 0.f, 0.f), -p.x);
		}
		if (p.x > box.x) {
			collisionFunc(particleSet, restitution, i, PCISPH::Vec3(-1.f, 0.f, 0.f), p.x - box.x);
		}
		if (p.y < 0) {
			collisionFunc(particleSet, restitution, i, PCISPH::Vec3(0.f, 1.f, 0.f), -p.y);
		}
		if (p.y > box.y) {
			collisionFunc(particleSet, restitution, i, PCISPH::Vec3(0.f, -1.f, 0.f), p.y - box.y);
		}
		if (p.z < 0) {
			collisionFunc(particleSet, restitution, i, PCISPH::Vec3(0.f, 0.f, 1.f), -p.z);
		}
		if (p.z > box.z) {
			collisionFunc(particleSet, restitution, i, PCISPH::Vec3(0.f, 0.f, -1.f), p.z - box.z);
		}
	}
}

void Simulator::initParticlesPositions() {
	float r = scene->particleRadius;
	float d = 2 * r;

	// initialize fluid particles
	PCISPH::Vec3 fBox = scene->fluidSize;
	PCISPH::Vec3 fPos = scene->fluidPosition;

	for (float x = r; x < fBox.x; x += d) {
		for (float y = r; y < fBox.y; y += d) {
			for (float z = r; z < fBox.z; z += d) {
				particleSet.addParticle(
					fPos + PCISPH::Vec3(x, y, z),				// position
					scene->initVelocity							// init velocity
				);
			}
		}
	}
}

void Simulator::initDensityVarianceScale() {
	float temp = scene->timeStep * particleSet.particleMass / scene->referenceDensity;
	float beta = 2.0f * temp * temp;
	PCISPH::Vec3 tmp1(0.0f);
	PCISPH::Vec3 tmp2(0.0f);
	float tmp3 = 0.0f;
	float r = scene->particleRadius;
	float d = 2.f * r;
	for (float x = -kernel.H - r; x <= kernel.H + r; x += d) {
		for (float y = -kernel.H - r; y <= kernel.H + r; y += d) {
			for (float z = -kernel.H - r; z <= kernel.H + r; z += d) {
				PCISPH::Vec3 pos(x, y, z);
				if (pos == PCISPH::Vec3(0.0f)) continue;

				PCISPH::Vec3 kernelValue1 = kernel.spikyKernelGradient(-pos);
				//PCISPH::Vec3 kernelValue1 = kernel.poly6KernelGradient(-pos);
				PCISPH::Vec3 kernelValue2 = kernel.poly6KernelGradient(-pos);
				tmp1 += kernelValue1;
				tmp2 += kernelValue2;
				tmp3 += PCISPH::dot(kernelValue1, kernelValue2);
			}
		}
	}
	this->densityVarianceScale = 1.0f / beta / (PCISPH::dot(tmp1, tmp2) + tmp3);
}

void Simulator::update(const size_t maxIterations) {


	updateFluidGrid();
	computeDensity<<<1, 1>>>(thrust::raw_pointer_cast(d_fluidGrid), thrust::raw_pointer_cast(particleSetPointer), thrust::raw_pointer_cast(offset.data()), thrust::raw_pointer_cast(d_kernel));
	computeNormal<<<1,1>>>(thrust::raw_pointer_cast(d_fluidGrid), thrust::raw_pointer_cast(particleSetPointer), thrust::raw_pointer_cast(offset.data()), thrust::raw_pointer_cast(d_kernel));
	computeForces<<<1,1>>>(thrust::raw_pointer_cast(d_fluidGrid), thrust::raw_pointer_cast(particleSetPointer), thrust::raw_pointer_cast(offset.data()), thrust::raw_pointer_cast(d_kernel), scene->referenceDensity, scene->viscosityCoefficient, scene->surfaceTensionCoefficient, scene->gravity);
	clearPressureAndPressureForce();
	size_t iterations = 0;
	while (iterations < maxIterations) {
		particleSet.maxDensityErr = 0.f;
		predictVelocityAndPosition<<<1,1>>>(thrust::raw_pointer_cast(particleSetPointer), scene->timeStep);
		updatePressure<<<1,1>>>(thrust::raw_pointer_cast(d_fluidGrid), thrust::raw_pointer_cast(particleSetPointer), thrust::raw_pointer_cast(offset.data()), thrust::raw_pointer_cast(d_kernel), scene->referenceDensity, densityVarianceScale);
		updatePressureForce<<<1,1>>>(thrust::raw_pointer_cast(d_fluidGrid), thrust::raw_pointer_cast(particleSetPointer), thrust::raw_pointer_cast(offset.data()), thrust::raw_pointer_cast(d_kernel));
		static const int MIN_ITERATIONS = 3;
		static const float yita = 0.01;
		static const float TOL = yita * this->scene->referenceDensity;
		if (++iterations >= MIN_ITERATIONS && this->particleSet.maxDensityErr < TOL) {
			break;
		}
	}

	updateVelocityAndPosition<<<1,1>>>(thrust::raw_pointer_cast(particleSetPointer), scene->timeStep);
	handleCollision<<<1,1>>>(thrust::raw_pointer_cast(particleSetPointer), scene->restitution, scene->boxSize);

	static size_t count = 0;
	std::cout << "maxDensityErr = " << this->particleSet.maxDensityErr << std::endl;
	std::cout << "iterations = " << iterations << std::endl;
	std::cout << "count = " << count << std::endl;
	std::cout << std::endl;
	count++;
}

