#include "hip/hip_runtime.h"
#include "Simulator.h"
#include "utils.h"

#include <iostream>

#include <thrust/device_new.h>

#define KERNEL_SCALE 4.0f
#define EPSILON 1e-30f

const bool DEBUG = false;

Simulator::Simulator() :particleSet(), scene(nullptr), kernel()
{
}


Simulator::~Simulator()
{
}

__global__ void initParticleSetPointer(
	DeviceParticleSetPointer* p,
	PCISPH::Vec3* position,
	PCISPH::Vec3* predictPosition,
	PCISPH::Vec3* velocity,
	PCISPH::Vec3* predictVelocity,
	PCISPH::Vec3* normal,
	PCISPH::Vec3* forces,
	PCISPH::Vec3* pressureForce,
	float* density,
	float* pressure,
	float particleMass,
	float maxDensityErr,
	size_t count
)
{
	p->position = position;
	p->predictPosition = predictPosition;
	p->velocity = velocity;
	p->predictVelocity = predictVelocity;
	p->normal = normal;
	p->forces = forces;
	p->pressureForce = pressureForce;
	p->density = density;
	p->pressure = pressure;
	p->particleMass = particleMass;
	p->maxDensityErr = maxDensityErr;
	p->count = count;
}

__global__ void initDeviceGrid(
	Grid* grid,
	float cellSize,
	PCISPH::Vec3 boxSize,
	PCISPH::uVec3 gridSize,
	size_t cellNumber
)
{
	grid->cellSize = cellSize;
	grid->boxSize = boxSize;
	grid->gridSize = gridSize;
	grid->cellNumber = cellNumber;
}

__global__
void initDeviceKernel(Kernel* k, float h)
{
	k->init(h);
}

void Simulator::init(const Scene *scene) {
	// Initialize scene
	this->scene = scene;

	// Initialize particleSet
	float particleMass = scene->referenceDensity * pow(2 * scene->particleRadius, 3);
	this->particleSet.init(particleMass);
	std::cout << "paricleMass = " << particleMass << std::endl;

	// Initialize fluid particles' and boundary particles' positions.
	this->initParticlesPositions();

	particleSetPointer = thrust::device_new<DeviceParticleSetPointer>();

	initParticleSetPointer << <1, 1 >> > (
		thrust::raw_pointer_cast(particleSetPointer),
		thrust::raw_pointer_cast(particleSet.position.data()),
		thrust::raw_pointer_cast(particleSet.predictPosition.data()),
		thrust::raw_pointer_cast(particleSet.velocity.data()),
		thrust::raw_pointer_cast(particleSet.predictVelocity.data()),
		thrust::raw_pointer_cast(particleSet.normal.data()),
		thrust::raw_pointer_cast(particleSet.forces.data()),
		thrust::raw_pointer_cast(particleSet.pressureForce.data()),
		thrust::raw_pointer_cast(particleSet.density.data()),
		thrust::raw_pointer_cast(particleSet.pressure.data()),
		particleSet.particleMass,
		particleSet.maxDensityErr,
		particleSet.count
		);

	hipDeviceSynchronize();
	// Initialize kernel
	float kernelRadius = KERNEL_SCALE * scene->particleRadius;
	kernel.init(kernelRadius);
	std::cout << "Kernel.H = " << kernel.H << std::endl;

	// Initialize grid
	this->fluidGrid.init(scene->boxSize, kernel.H);
	this->boundaryGrid.init(scene->boxSize, kernel.H);
	updateFluidGrid();


	// Initialize density variance scale
	this->initDensityVarianceScale();
	std::cout << "densityVarianceScale = " << this->densityVarianceScale << std::endl;

	offset.resize(fluidGrid.cellNumber + 1);


	d_fluidGrid = thrust::device_new<Grid>();
	initDeviceGrid << <1, 1 >> > (thrust::raw_pointer_cast(d_fluidGrid), fluidGrid.cellSize, fluidGrid.boxSize, fluidGrid.gridSize, fluidGrid.cellNumber);
	d_kernel = thrust::device_new<Kernel>();
	initDeviceKernel << <1, 1 >> > (thrust::raw_pointer_cast(d_kernel), kernelRadius);
	hipDeviceSynchronize();

	this->relax();

	std::cout << std::endl;
}
void Simulator::updateFluidGrid() {

	particleSet.h_velocity = particleSet.velocity;
	particleSet.h_position = particleSet.position;

	h_offset = offset;

	this->fluidGrid.update(particleSet.h_position, h_offset, [this](size_t i, size_t j) {
		std::swap(particleSet.h_position[i], particleSet.h_position[j]);
		std::swap(particleSet.h_velocity[i], particleSet.h_velocity[j]);
	});

	offset = h_offset;

	particleSet.position = particleSet.h_position;
	particleSet.velocity = particleSet.h_velocity;
}

__global__
void computeDensity(Grid* grid, DeviceParticleSetPointer* particles, size_t* offset, Kernel* kernel) {

	// compute fluid particles' densities
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particles->count) return;
	float fluidTerm = 0.0f;
	PCISPH::Vec3 pos = particles->position[i];

	PCISPH::uVec3 boundBoxMin = getGridPos(pos - PCISPH::Vec3(grid->cellSize), grid->boxSize, grid->cellSize);
	PCISPH::uVec3 boundBoxMax = getGridPos(pos + PCISPH::Vec3(grid->cellSize), grid->boxSize, grid->cellSize);

	for (auto z = boundBoxMin.z; z <= boundBoxMax.z; z++) {
		for (auto y = boundBoxMin.y; y <= boundBoxMax.y; y++) {
			for (auto x = boundBoxMin.x; x <= boundBoxMax.x; x++) {
				size_t cellIndex = grid->linearIndex(PCISPH::uVec3(x, y, z));
				for (size_t neighborIndex = offset[cellIndex]; neighborIndex < offset[cellIndex + 1]; neighborIndex++) {
					PCISPH::Vec3 r = pos - particles->position[neighborIndex];
					float rLen = PCISPH::length(r);
					if (rLen > kernel->H || rLen < EPSILON) continue;
					fluidTerm += kernel->poly6Kernel(r);
				}
			}
		}
	}

	particles->density[i] = particles->particleMass * fluidTerm;
}

__global__
void computeNormal(Grid* grid, DeviceParticleSetPointer* particles, size_t* offset, Kernel* kernel) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particles->count) return;

	PCISPH::Vec3 n(0.0f);

	glm::u64vec3 boundBoxMin = grid->getGridPos(particles->position[i] - PCISPH::Vec3(grid->cellSize));
	glm::u64vec3 boundBoxMax = grid->getGridPos(particles->position[i] + PCISPH::Vec3(grid->cellSize));

	for (auto z = boundBoxMin.z; z <= boundBoxMax.z; z++) {
		for (auto y = boundBoxMin.y; y <= boundBoxMax.y; y++) {
			for (auto x = boundBoxMin.x; x <= boundBoxMax.x; x++) {
				size_t cellIndex = grid->linearIndex(PCISPH::uVec3(x, y, z));
				for (size_t neighborIndex = offset[cellIndex]; neighborIndex < offset[cellIndex + 1]; neighborIndex++) {
					PCISPH::Vec3 r = particles->position[i] - particles->position[neighborIndex];
					float rLen = PCISPH::length(r);
					if (rLen > kernel->H || rLen < EPSILON) continue;
					n += kernel->poly6KernelGradient(r) / particles->density[neighborIndex];
				}
			}
		}
	}

	n *= kernel->H * particles->particleMass;
	particles->normal[i] = n;
}

__global__
void computeForces(Grid* grid, DeviceParticleSetPointer* particleSet, size_t* offset, Kernel* kernel,
	float referenceDensity, float viscosityCoefficient, float surfaceTensionCoefficient, PCISPH::Vec3 gravity) {
	float squaredMass = particleSet->particleMass * particleSet->particleMass;

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->count) return;

	PCISPH::Vec3 viscosity(0.f);
	PCISPH::Vec3 cohesion(0.f);
	PCISPH::Vec3 curvature(0.f);

	glm::u64vec3 boundBoxMin = grid->getGridPos(particleSet->position[i] - PCISPH::Vec3(grid->cellSize));
	glm::u64vec3 boundBoxMax = grid->getGridPos(particleSet->position[i] + PCISPH::Vec3(grid->cellSize));

	for (auto z = boundBoxMin.z; z <= boundBoxMax.z; z++) {
		for (auto y = boundBoxMin.y; y <= boundBoxMax.y; y++) {
			for (auto x = boundBoxMin.x; x <= boundBoxMax.x; x++) {
				size_t cellIndex = grid->linearIndex(PCISPH::uVec3(x, y, z));
				for (size_t j = offset[cellIndex]; j < offset[cellIndex + 1]; j++) {
					PCISPH::Vec3 r = particleSet->position[i] - particleSet->position[j];
					float rLen = PCISPH::length(r);
					if (rLen > kernel->H || rLen < EPSILON) continue;

					PCISPH::Vec3 vDiff = particleSet->velocity[i] - particleSet->velocity[j];
					viscosity -= vDiff * kernel->viscosityKernelLaplacian(r) / particleSet->density[j];

					float Kij = 2.0f * referenceDensity / (particleSet->density[i] + particleSet->density[j]);
					cohesion += Kij * (r / rLen) * kernel->cohesionKernel(rLen);
					curvature += Kij * (particleSet->normal[i] - particleSet->normal[j]);
				}
			}
		}
	}



	viscosity *= viscosityCoefficient * squaredMass / particleSet->density[i];
	cohesion *= -surfaceTensionCoefficient * squaredMass;
	curvature *= -surfaceTensionCoefficient * particleSet->particleMass;

	particleSet->forces[i] = viscosity + cohesion + curvature + particleSet->particleMass * gravity;
}

void Simulator::clearPressureAndPressureForce() {
	thrust::fill(particleSet.pressure.begin(), particleSet.pressure.end(), 0);
	thrust::fill(particleSet.pressureForce.begin(), particleSet.pressureForce.end(), PCISPH::Vec3(0, 0, 0));
}

__global__
void predictVelocityAndPosition(DeviceParticleSetPointer* particleSet, float timeStep) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->count) return;

	PCISPH::Vec3 acceleration = (particleSet->forces[i] + particleSet->pressureForce[i]) / particleSet->particleMass;
	particleSet->predictVelocity[i] = particleSet->velocity[i] + timeStep * acceleration;
	particleSet->predictPosition[i] = particleSet->position[i] + particleSet->predictVelocity[i] * timeStep;
}

__global__
void updatePressure(Grid* grid, DeviceParticleSetPointer* particleSet, size_t* offset, Kernel* kernel, float referenceDensity, float densityVarianceScale) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->count) return;



	float fDensity = 0.f;

	glm::u64vec3 boundBoxMin = grid->getGridPos(particleSet->position[i] - PCISPH::Vec3(grid->cellSize));
	glm::u64vec3 boundBoxMax = grid->getGridPos(particleSet->position[i] + PCISPH::Vec3(grid->cellSize));

	for (auto z = boundBoxMin.z; z <= boundBoxMax.z; z++) {
		for (auto y = boundBoxMin.y; y <= boundBoxMax.y; y++) {
			for (auto x = boundBoxMin.x; x <= boundBoxMax.x; x++) {
				size_t cellIndex = grid->linearIndex(PCISPH::uVec3(x, y, z));
				auto upper = offset[cellIndex + 1];
				for (size_t j = offset[cellIndex]; j < upper; j++) {
					PCISPH::Vec3 r = particleSet->predictPosition[i] - particleSet->predictPosition[j];
					float rLen = PCISPH::length(r);
					if (rLen > kernel->H || rLen < EPSILON) continue;
					fDensity += kernel->poly6Kernel(r);
				}
			}
		}
	}

	float bDensity = 0.f;

	float density = fDensity * particleSet->particleMass + bDensity;
	float densityVariation = max(0.f, density - referenceDensity);
	particleSet->maxDensityErr = max(particleSet->maxDensityErr, densityVariation);

	particleSet->pressure[i] += densityVarianceScale * densityVariation;
}

__global__
void updatePressureForce(Grid* grid, DeviceParticleSetPointer* particleSet, size_t* offset, Kernel* kernel) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->count) return;



	PCISPH::Vec3 pressureForce(0.f);

	glm::u64vec3 boundBoxMin = grid->getGridPos(particleSet->position[i] - PCISPH::Vec3(grid->cellSize));
	glm::u64vec3 boundBoxMax = grid->getGridPos(particleSet->position[i] + PCISPH::Vec3(grid->cellSize));


	for (auto z = boundBoxMin.z; z <= boundBoxMax.z; z++) {
		for (auto y = boundBoxMin.y; y <= boundBoxMax.y; y++) {
			for (auto x = boundBoxMin.x; x <= boundBoxMax.x; x++) {
				size_t cellIndex = grid->linearIndex(PCISPH::uVec3(x, y, z));
				for (size_t j = offset[cellIndex]; j < offset[cellIndex + 1]; j++) {
					PCISPH::Vec3 r = particleSet->predictPosition[i] - particleSet->predictPosition[j];
					float rLen = PCISPH::length(r);
					if (rLen > kernel->H || rLen < 1e-5f) continue;
					float term1 = particleSet->pressure[i] / particleSet->density[i] / particleSet->density[i];
					float term2 = particleSet->pressure[j] / particleSet->density[j] / particleSet->density[j];
					pressureForce -= particleSet->particleMass * (term1 + term2) * kernel->spikyKernelGradient(r);
				}
			}
		}
	}


	pressureForce *= particleSet->particleMass;
	particleSet->pressureForce[i] = pressureForce;
}

__global__
void updateVelocityAndPosition(DeviceParticleSetPointer* particleSet, float timeStep) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->count) return;



	PCISPH::Vec3 acceleration = (particleSet->forces[i] + particleSet->pressureForce[i]) / particleSet->particleMass;
	particleSet->velocity[i] += acceleration * timeStep;
	particleSet->position[i] += particleSet->velocity[i] * timeStep;
}

void Simulator::updateScene() {
	// TODO
}

void Simulator::relax() {
	update(10000);

	thrust::fill(particleSet.velocity.begin(), particleSet.velocity.end(), scene->initVelocity);
}

__device__
void collisionFunc(DeviceParticleSetPointer* particleSet, float restitution, size_t i, const PCISPH::Vec3 &n, const float d)
{
	particleSet->position[i] += d * n;
	particleSet->velocity[i] -= (1 + restitution) * PCISPH::dot(particleSet->velocity[i], n) * n;
}


__global__
void handleCollision(DeviceParticleSetPointer* particleSet, float restitution, PCISPH::Vec3 box) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= particleSet->count) return;


	const auto &p = particleSet->position[i];
	PCISPH::Vec3 n;
	if (p.x < 0) {
		collisionFunc(particleSet, restitution, i, PCISPH::Vec3(1.f, 0.f, 0.f), -p.x);
	}
	if (p.x > box.x) {
		collisionFunc(particleSet, restitution, i, PCISPH::Vec3(-1.f, 0.f, 0.f), p.x - box.x);
	}
	if (p.y < 0) {
		collisionFunc(particleSet, restitution, i, PCISPH::Vec3(0.f, 1.f, 0.f), -p.y);
	}
	if (p.y > box.y) {
		collisionFunc(particleSet, restitution, i, PCISPH::Vec3(0.f, -1.f, 0.f), p.y - box.y);
	}
	if (p.z < 0) {
		collisionFunc(particleSet, restitution, i, PCISPH::Vec3(0.f, 0.f, 1.f), -p.z);
	}
	if (p.z > box.z) {
		collisionFunc(particleSet, restitution, i, PCISPH::Vec3(0.f, 0.f, -1.f), p.z - box.z);
	}
}

void Simulator::initParticlesPositions() {
	float r = scene->particleRadius;
	float d = 2 * r;

	// initialize fluid particles
	PCISPH::Vec3 fBox = scene->fluidSize;
	PCISPH::Vec3 fPos = scene->fluidPosition;

	for (float x = r; x < fBox.x; x += d) {
		for (float y = r; y < fBox.y; y += d) {
			for (float z = r; z < fBox.z; z += d) {
				particleSet.addParticle(
					fPos + PCISPH::Vec3(x, y, z),				// position
					scene->initVelocity							// init velocity
				);
			}
		}
	}

	particleSet.predictPosition.resize(particleSet.count);
	thrust::fill(particleSet.predictPosition.begin(), particleSet.predictPosition.end(), PCISPH::Vec3(0.f));

	particleSet.predictVelocity.resize(particleSet.count);
	thrust::fill(particleSet.predictVelocity.begin(), particleSet.predictVelocity.end(), PCISPH::Vec3(0.f));

	particleSet.normal.resize(particleSet.count);
	thrust::fill(particleSet.normal.begin(), particleSet.normal.end(), PCISPH::Vec3(0.f));

	particleSet.forces.resize(particleSet.count);
	thrust::fill(particleSet.forces.begin(), particleSet.forces.end(), PCISPH::Vec3(0.f));

	particleSet.pressureForce.resize(particleSet.count);
	thrust::fill(particleSet.pressureForce.begin(), particleSet.pressureForce.end(), PCISPH::Vec3(0.f));

	particleSet.density.resize(particleSet.count);
	thrust::fill(particleSet.density.begin(), particleSet.density.end(), 0.f);

	particleSet.pressure.resize(particleSet.count);
	thrust::fill(particleSet.pressure.begin(), particleSet.pressure.end(), 0.f);

	particleSet.position = particleSet.h_position;
	particleSet.velocity = particleSet.h_velocity;

}

void Simulator::initDensityVarianceScale() {
	float temp = scene->timeStep * particleSet.particleMass / scene->referenceDensity;
	float beta = 2.0f * temp * temp;
	PCISPH::Vec3 tmp1(0.0f);
	PCISPH::Vec3 tmp2(0.0f);
	float tmp3 = 0.0f;
	float r = scene->particleRadius;
	float d = 2.f * r;
	for (float x = -kernel.H - r; x <= kernel.H + r; x += d) {
		for (float y = -kernel.H - r; y <= kernel.H + r; y += d) {
			for (float z = -kernel.H - r; z <= kernel.H + r; z += d) {
				PCISPH::Vec3 pos(x, y, z);
				if (pos == PCISPH::Vec3(0.0f)) return;

				PCISPH::Vec3 kernelValue1 = kernel.h_spikyKernelGradient(-pos);
				//PCISPH::Vec3 kernelValue1 = kernel.poly6KernelGradient(-pos);
				PCISPH::Vec3 kernelValue2 = kernel.h_poly6KernelGradient(-pos);
				tmp1 += kernelValue1;
				tmp2 += kernelValue2;
				tmp3 += PCISPH::dot(kernelValue1, kernelValue2);
			}
		}
	}
	this->densityVarianceScale = 1.0f / beta / (PCISPH::dot(tmp1, tmp2) + tmp3);
}

void Simulator::update(const size_t maxIterations) {

	int threadsPerBlock = 512;
	int numBlocks = (particleSet.count / threadsPerBlock) + 1;
	float maxDensityErr = 0;

	updateFluidGrid();
	computeDensity << <numBlocks, threadsPerBlock >> > (thrust::raw_pointer_cast(d_fluidGrid), thrust::raw_pointer_cast(particleSetPointer), thrust::raw_pointer_cast(offset.data()), thrust::raw_pointer_cast(d_kernel));
	hipDeviceSynchronize();
	computeNormal << <numBlocks, threadsPerBlock >> > (thrust::raw_pointer_cast(d_fluidGrid), thrust::raw_pointer_cast(particleSetPointer), thrust::raw_pointer_cast(offset.data()), thrust::raw_pointer_cast(d_kernel));
	hipDeviceSynchronize();
	computeForces << <numBlocks, threadsPerBlock >> > (thrust::raw_pointer_cast(d_fluidGrid), thrust::raw_pointer_cast(particleSetPointer), thrust::raw_pointer_cast(offset.data()), thrust::raw_pointer_cast(d_kernel), scene->referenceDensity, scene->viscosityCoefficient, scene->surfaceTensionCoefficient, scene->gravity);
	hipDeviceSynchronize();
	clearPressureAndPressureForce();
	size_t iterations = 0;
	while (iterations < maxIterations) {
		maxDensityErr = 0;
		hipMemcpy(&(this->particleSetPointer.get()->maxDensityErr), &maxDensityErr, sizeof(float), hipMemcpyHostToDevice);
		predictVelocityAndPosition << <numBlocks, threadsPerBlock >> > (thrust::raw_pointer_cast(particleSetPointer), scene->timeStep);
		hipDeviceSynchronize();
		updatePressure << <numBlocks, threadsPerBlock >> > (thrust::raw_pointer_cast(d_fluidGrid), thrust::raw_pointer_cast(particleSetPointer), thrust::raw_pointer_cast(offset.data()), thrust::raw_pointer_cast(d_kernel), scene->referenceDensity, densityVarianceScale);
		hipDeviceSynchronize();
		updatePressureForce << <numBlocks, threadsPerBlock >> > (thrust::raw_pointer_cast(d_fluidGrid), thrust::raw_pointer_cast(particleSetPointer), thrust::raw_pointer_cast(offset.data()), thrust::raw_pointer_cast(d_kernel));
		hipDeviceSynchronize();
		static const int MIN_ITERATIONS = 3;
		static const float yita = 0.01;
		static const float TOL = yita * this->scene->referenceDensity;
		hipMemcpy(&maxDensityErr, &(this->particleSetPointer.get()->maxDensityErr), sizeof(float), hipMemcpyDeviceToHost);
		if (++iterations >= MIN_ITERATIONS && maxDensityErr < TOL) {
			break;
		}
	}

	updateVelocityAndPosition << <numBlocks, threadsPerBlock >> > (thrust::raw_pointer_cast(particleSetPointer), scene->timeStep);
	hipDeviceSynchronize();
	handleCollision << <numBlocks, threadsPerBlock >> > (thrust::raw_pointer_cast(particleSetPointer), scene->restitution, scene->boxSize);
	hipDeviceSynchronize();


	static size_t count = 0;
	std::cout << "maxDensityErr = " << maxDensityErr << std::endl;
	std::cout << "iterations = " << iterations << std::endl;
	std::cout << "count = " << count << std::endl;
	std::cout << std::endl;
	count++;
}

