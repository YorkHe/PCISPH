#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <thrust/detail/config/host_device.h>

#include "Grid.h"
#include "utils.h"
Grid::Grid() :cellSize() {

}

Grid::~Grid() {

}

void Grid::init(const PCISPH::Vec3 boxSize, float cellSize) {
	this->cellSize = cellSize;
	this->boxSize = boxSize;

	this->gridSize = PCISPH::iVec3(
		(int)floor(boxSize.x / cellSize) + 1,
		(int)floor(boxSize.y / cellSize) + 1,
		(int)floor(boxSize.z / cellSize) + 1
	);
	cellNumber = gridSize.x * gridSize.y * gridSize.z;

}

void Grid::update(const thrust::host_vector<PCISPH::Vec3>& positions, thrust::host_vector<size_t>& offset, std::function<void(size_t, size_t)> swap){
	// particle count in each grid cell
	std::vector<size_t> cellCount(cellNumber, 0);
	// pointer to particle array index
	std::vector<size_t> cellIndex(cellNumber, 0);

	size_t particleNumber = positions.size();
	// grid indices for each particle
	std::vector<size_t> indices(particleNumber);

#pragma omp parallel for
	for (size_t i = 0; i < particleNumber; i++) {
		size_t index = linearIndex(positions[i]);
		indices[i] = index;
		cellCount[index] += 1;
	}
	offset.resize(cellNumber);

	size_t index = 0;
#pragma omp parallel for
	for (size_t i = 0; i < cellNumber; i++) {
		offset[i] = index;
		cellIndex[i] = index;
		index += cellCount[i];
	}

	auto a = offset.data();
	
	offset.back() = index;

#pragma omp parallel for
	for (size_t i = 0; i < particleNumber; i++) {
		while (i < offset[indices[i]] || i >= cellIndex[indices[i]]) {
			size_t j = cellIndex[indices[i]]++;
			std::swap(indices[i], indices[j]);
			swap(i, j);
		}
	}
}

PCISPH::uVec3 Grid::getGridPos(const PCISPH::Vec3 pos) const {
	PCISPH::Vec3 p = pos;

	if (p.x < 0) p.x = 0;
	if (p.y < 0) p.y = 0;
	if (p.z < 0) p.z = 0;

	if (p.x > boxSize.x) p.x = boxSize.x;
	if (p.y > boxSize.y) p.y = boxSize.y;
	if (p.z > boxSize.z) p.z = boxSize.z;


	return PCISPH::uVec3(
		(PCISPH::uint)floor(p.x / cellSize),
		(PCISPH::uint)floor(p.y / cellSize),
		(PCISPH::uint)floor(p.z / cellSize)
	);
}

__device__ __host__ 
PCISPH::uVec3 getGridPos(const PCISPH::Vec3 &pos, PCISPH::Vec3 boxSize, float cellSize) {
	PCISPH::Vec3 p = pos;

	if (p.x < 0) p.x = 0;
	if (p.y < 0) p.y = 0;
	if (p.z < 0) p.z = 0;

	if (p.x > boxSize.x) p.x = boxSize.x;
	if (p.y > boxSize.y) p.y = boxSize.y;
	if (p.z > boxSize.z) p.z = boxSize.z;

	return PCISPH::uVec3(
		(PCISPH::uint)floor(p.x / cellSize),
		(PCISPH::uint)floor(p.y / cellSize),
		(PCISPH::uint)floor(p.z / cellSize)
	);
}


__device__ __host__ 
size_t Grid::linearIndex(const PCISPH::uVec3 gridPos) const {
	return gridPos.x + gridPos.y * gridSize.x + gridPos.z * gridSize.x * gridSize.y;
}

__device__ __host__ 
size_t Grid::linearIndex(const PCISPH::Vec3 pos) const {
	return linearIndex(getGridPos(pos));
}

/* Query the neighbor points
* Then do func(i, j)
* Expect Func format:
* void Func(size_t j);
*/
//template<typename Func>
